#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"
#include "stdio.h"
#include "cuda_debug.h"
#include "../../../constants.h"


#define BIT(x,i) ((x & gcuBitMask[i]) >> 2*i)

// Ternary Gates
int gTernaryOps[5][3]= 
{
  {2, 0, 1},      // 0: -1
  {1, 2, 0},      // 1: -2
  {1, 0, 2},      // 2: 01
  {2, 1, 0},      // 3: 02
  {0, 2, 1}       // 4: 12
};                

// Operation to use based on [input][output] values which are an index to the gates in the m_Op array above
// Example: 
//    Input = 2, Output = 1 => Gate 4 (From m_Op above would be Swap gate 12)
int gOpMap[3][3] =
{
  // Output    0, 1, 2     Input 
  {4, 2, 3}, // 0
  {2, 3, 4}, // 1
  {3, 4, 2}  // 2
};

// Bitmask two bits at a time for ternary operations.
int gBitMask[] = {3, 3<<2, 3<<4, 3<<6, 3<<8, 3<<10, 3<<12, 3<<14, 3<<16}; 
__device__ __constant__ int gcuBitMask[sizeof(gBitMask)];
__device__ __constant__ int gcuTernaryOps[5][3];
__device__ __constant__ int gcuOpMap[3][3];

__device__ void Process(int inTerm, int outTerm, int nBits, PINT gBitMask, PINT pControl, PBYTE pTarget, PBYTE pOperation);


__device__ void CopySharedToGlobal(PINT pDst, PINT pSrc, int nWords)
{
  for (int i=0; i<nWords; i++)
    pDst[i] = pSrc[i];
}

__device__ void CopySharedBytesToGlobal(PBYTE pDst, PBYTE pSrc, int nBytes)
{
  for (int i=0; i<nBytes; i++)
    pDst[i] = pSrc[i];
}

__global__ void cuSynthesizeKernel(CudaSequence *data)
{
  CudaSequence seq = data[0];
  int inputIndex =  threadIdx.x * seq.m_nTerms; 
  int outputIndex = threadIdx.x * seq.m_nMaxGates;
  int nGates = 0;
  int nBits = seq.m_nBits;

  __shared__ int pIn[243];
  __shared__ int pOut[243];
  __shared__ int pControl[3*1024];
  __shared__ BYTE pGates[3*1024];
  __shared__ BYTE pTarget[3*1024];

  for(int i=0; i<seq.m_nTerms; i++) {
    pIn[i] = seq.m_cuIn[inputIndex+i]; 
    pOut[i] = seq.m_cuOut[inputIndex+i]; 
  }

  for(int i=0; i<seq.m_nTerms; i++) {
    Process(pIn[i], 
      pOut[i], 
      nBits,
      &nGates,
      pControl,
      pTarget,
      pGates
    );
  }

  __syncthreads();

  for(int i=0; i<nGates; i++) {
    CopySharedToGlobal(&seq.m_cuControl[outputIndex], pControl, nGates);
    CopySharedBytesToGlobal(&seq.m_cuTarget[outputIndex], pTarget, nGates);
    CopySharedBytesToGlobal(&seq.m_cuGates[outputIndex], pGates, nGates);
  }
  seq.m_cuNumGates[threadIdx.x] = nGates;

  //printf("block: nGates Index: %d [%d]\n", blockIdx.x, seq.m_cuGates[blockIdx.x]);
}


void SynthesizeKernel(CudaSequence *pcuSeq, int nSequences)
{
  // Constants are scoped to a file, and cannot use extern..
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuBitMask), gBitMask, sizeof(gBitMask)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuTernaryOps), gTernaryOps, sizeof(gTernaryOps)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuOpMap), gOpMap, sizeof(gOpMap)) );
  cuSynthesizeKernel<<<1, nSequences>>>(pcuSeq);
}

__device__ int Propagate(int outTerm, PINT pControl, PBYTE pTarget, PBYTE pOperation, int nGates)
{
  // Apply current list of gates..
  for (int i=0; i<nGates; i++) {
    int mask = gcuBitMask[pTarget[i]];
    if ( pControl[i] == (~mask & outTerm) ) {               // Control Bits for gate matches All bits in output excluding target bits.
      int val = (mask & outTerm) >> 2*pTarget[i];           // Bring target bits to lower two bits.
      val = (gcuTernaryOps[pOperation[i]][val] << 2*pTarget[i]);       // Apply operation on bits.
      outTerm = (~mask & outTerm) | val;
    }
  }

  return outTerm;
}

__device__ void Process(int inTerm, int outTerm, int nBits, PINT pnGates, PINT pControl, PBYTE pTarget,  PBYTE pOperation)
{
  //printf("\n****** In,out:[%d, %d], nGates: [%d] ", inTerm, outTerm, *pnGates);
  outTerm = Propagate(outTerm, pControl, pTarget, pOperation, *pnGates);

  //printf("After Propgate: %d \n", outTerm);

  //  process low (output) to high (input) transitions first then high to low
  for(int dir=1; dir>-2; dir-=2) {
    for (int i=0; i < nBits; i++) {
      // Isloate bit (i) for processing
      int inBit  = (gcuBitMask[i] & inTerm);        // must be defined as signed int
      int outBit = (gcuBitMask[i] & outTerm);

      if ( dir * (inBit - outBit) > 0) {         // Difference? Yes!
        pTarget   [*pnGates] = i;                           // Save index of target bits
        pControl  [*pnGates] = ~gcuBitMask[i] & outTerm;      // For now, it is everything except target bits is a control bit
        pOperation[*pnGates] = gcuOpMap[BIT(inTerm,i)][BIT(outTerm,i)];  // Find the appropriate operation. 
       // printf("dir(%d) [C,T,O] [%d, %d, %d] ", dir, pControl[*pnGates], i, pOperation[*pnGates]);
        (*pnGates)++;
        outTerm = (~gcuBitMask[i] & outTerm) | (gcuBitMask[i] & inTerm);
       // printf(" => %d \n", outTerm);
      }
    }
  }
}