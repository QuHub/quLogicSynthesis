#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"
#include "stdio.h"
#include "cuda_debug.h"


#define BIT(x,i) ((x & gcuBitMask[i]) >> 2*i)

// Ternary Gates
int gTernaryOps[5][3]= 
{
  {2, 0, 1},      // 0: -1
  {1, 2, 0},      // 1: -2
  {1, 0, 2},      // 2: 01
  {2, 1, 0},      // 3: 02
  {0, 2, 1}       // 4: 12
};                

// Operation to use based on [input][output] values which are an index to the gates in the m_Op array above
// Example: 
//    Input = 2, Output = 1 => Gate 4 (From m_Op above would be Swap gate 12)
int gOpMap[3][3] =
{
  // Output    0, 1, 2     Input 
  {4, 2, 3}, // 0
  {2, 3, 4}, // 1
  {3, 4, 2}  // 2
};

// Bitmask two bits at a time for ternary operations.
int gBitMask[] = {3, 3<<2, 3<<4, 3<<6, 3<<8, 3<<10, 3<<12, 3<<14, 3<<16}; 
__device__ __constant__ int gcuBitMask[sizeof(gBitMask)];
__device__ __constant__ int gcuTernaryOps[5][3];
__device__ __constant__ int gcuOpMap[3][3];

__device__ void Process(int inTerm, int outTerm, int nBits, PINT gBitMask, PINT pControl, PBYTE pTarget, PBYTE pOperation);


__global__ void cuSynthesizeKernel(CudaSequence *data)
{
  CudaSequence seq = data[0];
  int inputIndex =  blockIdx.x * seq.m_nTerms; 
  int outputIndex = blockIdx.x * seq.m_nMaxGates; 
  seq.m_cuGates[blockIdx.x] = 0;

  for(int i=0; i<seq.m_nTerms; i++) {
    Process(seq.m_cuIn[inputIndex+i], 
      seq.m_cuOut[inputIndex+i], 
      seq.m_nBits,
      &seq.m_cuGates[blockIdx.x],
      &seq.m_cuControl[outputIndex],
      &seq.m_cuTarget[outputIndex], 
      &seq.m_cuOperation[outputIndex]
    );
  }
  //printf("block: nGates Index: %d [%d]\n", blockIdx.x, seq.m_cuGates[blockIdx.x]);
}

void SynthesizeKernel(CudaSequence *pcuSeq)
{
  // Constants are scoped to a file, and cannot use extern..
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuBitMask), gBitMask, sizeof(gBitMask)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuTernaryOps), gTernaryOps, sizeof(gTernaryOps)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuOpMap), gOpMap, sizeof(gOpMap)) );
  cuSynthesizeKernel<<<NUMBER_OF_CUDA_BLOCKS, 1>>>(pcuSeq);
}

__device__ int Propagate(int outTerm, PINT pControl, PBYTE pTarget, PBYTE pOperation, int nGates)
{
  // Apply current list of gates..
  for (int i=0; i<nGates; i++) {
    int mask = gcuBitMask[pTarget[i]];
    if ( pControl[i] == (~mask & outTerm) ) {               // Control Bits for gate matches All bits in output excluding target bits.
      int val = (mask & outTerm) >> 2*pTarget[i];           // Bring target bits to lower two bits.
      val = (gcuTernaryOps[pOperation[i]][val] << 2*pTarget[i]);       // Apply operation on bits.
      outTerm = (~mask & outTerm) | val;
    }
  }

  return outTerm;
}

__device__ void Process(int inTerm, int outTerm, int nBits, PINT pnGates, PINT pControl, PBYTE pTarget,  PBYTE pOperation)
{
  //printf("\n****** In,out:[%d, %d] ", inTerm, outTerm);
  outTerm = Propagate(outTerm, pControl, pTarget, pOperation, *pnGates);

  //printf("After Propgate: %d \n", outTerm);

  //  process low (output) to high (input) transitions first then high to low
  for(int dir=1; dir>-2; dir-=2) {
    for (int i=0; i < nBits; i++) {
      // Isloate bit (i) for processing
      int inBit  = (gcuBitMask[i] & inTerm);        // must be defined as signed int
      int outBit = (gcuBitMask[i] & outTerm);

      if ( dir * (inBit - outBit) > 0) {         // Difference? Yes!
        pTarget   [*pnGates] = i;                           // Save index of target bits
        pControl  [*pnGates] = ~gcuBitMask[i] & outTerm;      // For now, it is everything except target bits is a control bit
        pOperation[*pnGates] = gcuOpMap[BIT(inTerm,i)][BIT(outTerm,i)];  // Find the appropriate operation. 
        //printf("dir(%d) [C,T,O] [%d, %d, %d] ", dir, pControl[*pnGates], i, pOperation[*pnGates]);
        (*pnGates)++;
        outTerm = (~gcuBitMask[i] & outTerm) | (gcuBitMask[i] & inTerm);
        //printf(" => %d \n", outTerm);
      }
    }
  }
}