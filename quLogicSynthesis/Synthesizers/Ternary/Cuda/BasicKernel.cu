#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"
#include "stdio.h"
#include "cuda_debug.h"


#define BIT(x,i) ((x & gcuBitMask[i]) >> 2*i)

// Ternary Gates
int gTernaryOps[5][3]= 
{
  {2, 0, 1},      // 0: -1
  {1, 2, 0},      // 1: -2
  {1, 0, 2},      // 2: 01
  {2, 1, 0},      // 3: 02
  {0, 2, 1}       // 4: 12
};                

// Operation to use based on [input][output] values which are an index to the gates in the m_Op array above
// Example: 
//    Input = 2, Output = 1 => Gate 4 (From m_Op above would be Swap gate 12)
int gOpMap[3][3] =
{
  // Output    0, 1, 2     Input 
  {4, 2, 3}, // 0
  {2, 3, 4}, // 1
  {3, 4, 2}  // 2
};

// Bitmask two bits at a time for ternary operations.
int gBitMask[] = {3, 3<<2, 3<<4, 3<<6, 3<<8, 3<<10, 3<<12, 3<<14, 3<<16}; 
__device__ __constant__ int gcuBitMask[sizeof(gBitMask)];
__device__ __constant__ int gcuTernaryOps[5][3];
__device__ __constant__ int gcuOpMap[3][3];

__device__ void Process(int inTerm, int outTerm, int nBits, PINT gBitMask, PINT pTarget, PINT pControl, PINT pOperation);


__global__ void cuSynthesizeKernel(CudaSequence *data)
{
  CudaSequence seq = data[0];
  int index = blockIdx.x * seq.m_outputBlockSize;
  seq.m_cuGates[index] = 0;

  for(int i=0; i<seq.m_nTerms; i++) {
    Process(seq.m_cuIn[index+i], 
      seq.m_cuOut[index+i], 
      seq.m_nBits,
      &seq.m_cuGates[index],
      &seq.m_cuTarget[index], 
      &seq.m_cuControl[index],
      &seq.m_cuOperation[index]
    );
  }

  //for(int i=0; i < 10; i++) {
  //  target[index+i] = 0x20;
  //  control[index+i] = index*100;
  //}

}

void SynthesizeKernel(CudaSequence *pcuSeq)
{
  // Constants are scoped to a file, and cannot use extern..
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuBitMask), gBitMask, sizeof(gBitMask)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuTernaryOps), gTernaryOps, sizeof(gTernaryOps)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuOpMap), gOpMap, sizeof(gOpMap)) );
  cuSynthesizeKernel<<<1, 1>>>(pcuSeq);
}

__device__ int Propagate(int outTerm, PINT pTarget, PINT pOperation, PINT pControl, int nGates)
{
  // Apply current list of gates..
  for (int i=0; i<nGates; i++) {
    int mask = gcuBitMask[pTarget[i]];
    if ( pControl[i] == (~mask & outTerm) ) {               // Control Bits for gate matches All bits in output excluding target bits.
      int val = (mask & outTerm) >> 2*pTarget[i];           // Bring target bits to lower two bits.
      val = (gcuTernaryOps[pOperation[i]][val] << 2*pTarget[i]);       // Apply operation on bits.
      outTerm = (~mask & outTerm) | val;
    }
  }

  return outTerm;
}

__device__ void Process(int inTerm, int outTerm, int nBits, PINT pnGates, PINT pTarget, PINT pControl, PINT pOperation)
{
  printf("\n****** In,out:[%d, %d] ", inTerm, outTerm);
  outTerm = Propagate(outTerm, pTarget, pOperation, pControl, *pnGates);

  printf("After Propgate: %d \n", outTerm);

  //  process low (output) to high (input) transitions first then high to low
  for(int dir=1; dir>-2; dir-=2) {
    for (int i=0; i < nBits; i++) {
      // Isloate bit (i) for processing
      int inBit  = (gcuBitMask[i] & inTerm);        // must be defined as signed int
      int outBit = (gcuBitMask[i] & outTerm);

      if ( dir * (inBit - outBit) > 0) {         // Difference? Yes!
        pTarget   [*pnGates] = i;                           // Save index of target bits
        pControl  [*pnGates] = ~gcuBitMask[i] & outTerm;      // For now, it is everything except target bits is a control bit
        pOperation[*pnGates] = gcuOpMap[BIT(inTerm,i)][BIT(outTerm,i)];  // Find the appropriate operation. 
        printf("dir(%d) [C,T,O] [%d, %d, %d] ", dir, pControl[*pnGates], i, pOperation[*pnGates]);
        (*pnGates)++;
        outTerm = (~gcuBitMask[i] & outTerm) | (gcuBitMask[i] & inTerm);
        printf(" => %d \n", outTerm);
      }
    }
  }
}


